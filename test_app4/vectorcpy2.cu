
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void VecCpy(float* A,float *B,float* C,int N)
{
	int i=blockDim.x*blockIdx.x+threadIdx.x;
	float c=0;
	float a=A[i];
	float b=B[i];
	if(i+10<N){
		for(int p=0;p<100;p++)
			c+=a*b;
		for(int t=0;t<10;t++)
			C[i+t] = A[i+t];
		// C[i]=A[i];
		// C[i+1]=A[i+1];
		// C[i+2]=A[i+2];
		// C[i+3]=A[i+3];
		// C[i+4]=A[i+4];
		// C[i+5]=A[i+5];
		// C[i+6]=A[i+6];
		// C[i+7]=A[i+7];
		// C[i+8]=A[i+8];
		// C[i+9]=A[i+9];
		C[i]=c;
	}
}


void RandomInit(float* data, int n)
{
    for (int i = 0; i < n; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

int main(){
	int N=32*20;
	size_t size=N*sizeof(float);

	float* h_A=(float*)malloc(size);
	float* h_B=(float*)malloc(size);
	float* h_C=(float*)malloc(size);
	
	RandomInit(h_A,N);
	RandomInit(h_B,N);

	float*d_A;
	hipMalloc(&d_A,size);
	float*d_B;
	hipMalloc(&d_B,size);
	float*d_C;
	hipMalloc(&d_C,size);
	
	hipMemcpy(d_A,h_A,size,hipMemcpyHostToDevice);
	hipMemcpy(d_B,h_B,size,hipMemcpyHostToDevice);	
	int threadsPerBlock=32*20;
	int blocksPerGrid=(N+threadsPerBlock-1)/threadsPerBlock;
	VecCpy<<<blocksPerGrid, threadsPerBlock>>>(d_A,d_B,d_C,N);
	hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost);

	for(int i=0; i<N; i++)
		printf("%f \n",h_C[i]);
	
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	
	free(h_A);
	free(h_B);
	free(h_C);
	
}

