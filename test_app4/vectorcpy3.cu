
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void VecCpy(float* A,float *B,float* C,float *D,int N)
{
	int i=blockDim.x*blockIdx.x+threadIdx.x;
	float c=0;
	float a=A[i];
	float b=B[i];
	__syncthreads();
//	if(i+10<N){
	B[i]=D[i];
	for(int p=0;p<100;p++)
		c+=a+b;
		// for(int t=0;t<10;t++)
		// 	C[i+t] = A[i+t];
		// C[i]=A[i];
		// C[i+1]=A[i+1];
		// C[i+2]=A[i+2];
		// C[i+3]=A[i+3];
		// C[i+4]=A[i+4];
		// C[i+5]=A[i+5];
		// C[i+6]=A[i+6];
		// C[i+7]=A[i+7];
		// C[i+8]=A[i+8];
		// C[i+9]=A[i+9];
	C[i]=c;
}


void RandomInit(float* data, int n)
{
    for (int i = 0; i < n; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

int main(){
	int N=32*48;
	size_t size=N*sizeof(float);

	float* h_A=(float*)malloc(size);
	float* h_B=(float*)malloc(size);
	float* h_C=(float*)malloc(size);
	float* h_D=(float*)malloc(size);
	
	RandomInit(h_A,N);
	RandomInit(h_B,N);
	RandomInit(h_D,N);

	float*d_A;
	hipMalloc(&d_A,size);
	float*d_B;
	hipMalloc(&d_B,size);
	float*d_C;
	hipMalloc(&d_C,size);
	float*d_D;
	hipMalloc(&d_D,size);
	
	hipMemcpy(d_A,h_A,size,hipMemcpyHostToDevice);
	hipMemcpy(d_B,h_B,size,hipMemcpyHostToDevice);
	hipMemcpy(d_D,h_D,size,hipMemcpyHostToDevice);
	int threadsPerBlock=32*48;
	int blocksPerGrid=(N+threadsPerBlock-1)/threadsPerBlock;
	VecCpy<<<blocksPerGrid, threadsPerBlock>>>(d_A,d_B,d_C,d_D,N);
	hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost);
	hipMemcpy(h_B,d_B,size,hipMemcpyDeviceToHost);

	for(int i=0; i<N; i++)
		printf("%f \n",h_C[i]);
	
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	hipFree(d_D);
	
	free(h_A);
	free(h_B);
	free(h_C);
	free(h_D);
	
}

