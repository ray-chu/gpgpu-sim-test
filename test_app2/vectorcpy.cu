
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void VecCpy(float* A,float* C,int N)
{
	int i=blockDim.x*blockIdx.x+threadIdx.x;
	if(i<N)
		C[i]=A[i];
}


void RandomInit(float* data, int n)
{
    for (int i = 0; i < n; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

int main(){
	int N=38400;
	size_t size=N*sizeof(float);

	float* h_A=(float*)malloc(size);
	float* h_C=(float*)malloc(size);
	
	RandomInit(h_A,N);

	float*d_A;
	hipMalloc(&d_A,size);
	float*d_C;
	hipMalloc(&d_C,size);
	
	hipMemcpy(d_A,h_A,size,hipMemcpyHostToDevice);
	int threadsPerBlock=320;
	int blocksPerGrid=(N+threadsPerBlock-1)/threadsPerBlock;
	VecCpy<<<blocksPerGrid, threadsPerBlock>>>(d_A,d_C,N);
	hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost);

	for(int i=0; i<N; i++)
		printf("%f \n",h_C[i]);
	
	hipFree(d_A);
	hipFree(d_C);
	
	free(h_A);
	free(h_C);
	
}

